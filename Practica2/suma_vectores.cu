
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>

#define THREADS_PER_BLOCK 1024

__global__ void add(int n, float *x, float *y) {

    int i = THREADS_PER_BLOCK * blockIdx.x + threadIdx.x;

    if (i < n){
        y[i] += x[i];
    }
}

int main(void) {
    
    int N = 1 << 20;  // N = 2^20 = 1024*1024= 1.048.576
    int N_blocks = 1 + (N-1)/THREADS_PER_BLOCK; // ceiling(N/THREADS_PER_BLOCK)
    float *x; // = new float[N];
    float *y; // = new float[N]; 
    hipMallocManaged(&x, N*sizeof(float));
    hipMallocManaged(&y, N*sizeof(float));
    
    for (int i = 0; i < N; i++){
        x[i]= 1.0f;
        y[i]= 2.0f;
    }
    add<<<N_blocks,THREADS_PER_BLOCK>>>(N, x, y);
    hipDeviceSynchronize();
    float maxError = 0.0f;
    int contError = 0;

    for (int i = 0; i < N; i++){
       maxError = fmax(maxError,fabs(y[i]-3.0f));
       if (y[i] != 3.0) contError++; 
    }
    std::cout << "Suma de " << N << " elementos" << std::endl;
    std::cout << "Número de errores: " <<contError << std::endl;
    std::cout << "Max error: " <<maxError << std::endl;

    hipFree (x);
    hipFree (y);
   
   return 0;
}