
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>


__global__ void add(int n, float *x, float *y) {
    
    for (int i =0; i < n; i++){
        y[i] = x[i]+y[i];
    }
}

int main(void) {
    
    int N = 1 << 20;  // N = 2^20 = 1024*1024= 1.048.576
    float *x; // = new float[N];
    float *y; // = new float[N]; 
    hipMallocManaged(&x, N*sizeof(float));
    hipMallocManaged(&y, N*sizeof(float));
    
    for (int i = 0; i < N; i++){
        x[i] = 1.0f;
        y[i] = 2.0f;
    }
    add<<<1,1>>>(N, x, y);
    hipDeviceSynchronize();
    float maxError = 0.0f;
    int contError = 0;

    for (int i = 0; i < N; i++){
       maxError = fmax(maxError, fabs(y[i]-3.0f));
       if (y[i] != 3.0) contError++; 
    }
    std::cout << "suma de " << N << " Elementos" << std::endl;
    std::cout << "Número de Errores: " <<contError << std::endl;
    std::cout << "Max error: " <<maxError << std::endl;
    hipFree (x);
    hipFree (y);
   
   return 0;
}
