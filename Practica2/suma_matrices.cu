
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>

#define BLOCK_SIZE 32

__global__ void add(int n, float **x, float **y) {

    int i = BLOCK_SIZE * blockIdx.x + threadIdx.x;
    int j = BLOCK_SIZE * blockIdx.y + threadIdx.y;

    if (i < n && j < n){
        y[i][j] += x[i][j];
    }
}

int main(void) {
    
    int N = 1 << 10;  // N = 2^10 = 1024
    int N_blocks = 1 + (N-1)/BLOCK_SIZE;
    dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
    dim3 blocks(N_blocks, N_blocks);
    float **x;
    float **y;

    hipMallocManaged(&x, N*sizeof(float *));
    hipMallocManaged(&y, N*sizeof(float *));

    for (int i = 0; i < N; i++){
        hipMallocManaged(x+i, N*sizeof(float));
        hipMallocManaged(y+i, N*sizeof(float));
    }
    
    for (int i = 0; i < N; i++){
        for (int j = 0; j < N; j++){
            x[i][j] = 1.0f;
            y[i][j] = 2.0f;
        }
    }
    add<<<blocks,threads>>>(N, x, y);
    hipDeviceSynchronize();
    float maxError = 0.0f;
    int contError = 0;

    for (int i = 0; i < N; i++){
        for (int j = 0; j < N; j++){
            maxError = fmax(maxError,fabs(y[i][j]-3.0f));
            if (y[i][j] != 3.0) contError++;
        } 
    }
    std::cout << "Suma de " << N << "x" << N << " elementos" << std::endl;
    std::cout << "Número de errores: " << contError << std::endl;
    std::cout << "Max error: " << maxError << std::endl;

    for (int i = 0; i < N; i++){
        hipFree(x[i]);
        hipFree(y[i]);
    }

    hipFree (x);
    hipFree (y);
   
   return 0;
}