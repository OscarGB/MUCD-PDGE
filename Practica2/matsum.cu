#include "hip/hip_runtime.h"
#include "../common/book.h"
#define N 10

__global__ void add( int *a, int *b, int *c ) {
	int tid = blockIdx.x; // handle the data at this index
	if (tid < N)
	c[tid] = a[tid] + b[tid];
}

int main( void ) {
	int a[N], b[N], c[N];
	int *dev_a, *dev_b, *dev_c;
	// allocate the memory on the GPU
	HANDLE_ERROR( hipMalloc( (void**)&dev_a, N * sizeof(int) ) );
	HANDLE_ERROR( hipMalloc( (void**)&dev_b, N * sizeof(int) ) );
	HANDLE_ERROR( hipMalloc( (void**)&dev_c, N * sizeof(int) ) );
	// fill the arrays 'a' and 'b' on the CPU
	for (int i=0; i<N; i++) {
		a[i] = -i;
		b[i] = i * i;
	}