
#include <hip/hip_runtime.h>
#include <stdio.h>
#define N 16
#define BLOCK_SIZE 32 < N ? 32 : N
void matrixMultCPU(int a[N][N], int b[N][N], int c[N][N]) {
	int n,m;
	for (int i = 0; i < N; i++) {
		for (int j = 0; j < N; j++) {
			int sum = 0;
			for (int k = 0; k < N; k++) {
				m = a[i][k];
				n = b[k][j];
				sum += m * n;
			}
			c[i][j] = sum;
		}
	}
}


__global__ void matrixMultGPU(int *a, int *b, int *c) {
	int k, sum = 0;
	int col = threadIdx.x + blockDim.x * blockIdx.x;
	int fil = threadIdx.y + blockDim.y * blockIdx.y;

	__shared__ float A[BLOCK_SIZE][N];
	__shared__ float B[BLOCK_SIZE][N];

	for (int i = threadIdx.x; i < N; i+=blockDim.x){
		A[threadIdx.y][i] = a[fil*N + i];
	}
	for (int i = threadIdx.y; i < N; i+=blockDim.y){
		B[threadIdx.x][i] = b[i*N + col];
	}
	
	__syncthreads();

	if (col < N && fil < N) {
		// #pragma unroll
		for (k = 0; k < N; k++) {
			sum += A[threadIdx.y][k] * B[threadIdx.x][k];
		}
		c[fil * N + col] = sum;
	}
}


int main() {
	int a[N][N], b[N][N], c[N][N], d[N][N];
	int *dev_a, *dev_b, *dev_c;
	int cont,i,j;
/* inicializando variables con datos*/
	for (i = 0; i < N; i++) {
		cont = 0;
		for (j = 0; j < N; j++) {
			a[i][j] = cont;
			b[i][j] = cont;
			cont++;
		}
	}
	int size = N * N * sizeof(int);

	hipMalloc((void **) &dev_a, size);
	hipMalloc((void **) &dev_b, size);
	hipMalloc((void **) &dev_c, size);

	hipMemcpy(dev_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, size, hipMemcpyHostToDevice);

	dim3 dimGrid((N+32-1)/32, (N+32-1)/32);
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);

  	// Allocate CUDA events that we'll use for timing
	hipEvent_t start;
	hipEventCreate(&start);
	hipEvent_t stop;
	hipEventCreate(&stop);
 // Record the start event
	hipEventRecord(start, NULL);
// Repita la ejecucion del kernel 1000 veces para eliminar
// efectos de arranque en frio
	int nIter = 1000;
	for (int j = 0; j < nIter; j++)
		matrixMultGPU<<<dimGrid, dimBlock>>>(dev_a, dev_b, dev_c);

 // Record the stop event
	hipEventRecord(stop, NULL);
 // Wait for the stop event to complete
	hipEventSynchronize(stop);
	float msecTotal = 0.0f;
	hipEventElapsedTime(&msecTotal, start, stop);
// Compute and print the performance
	float msecPerKernelExecution = msecTotal / nIter;
	double flopsPerMMul = 2.0 * N * N * N;
	double gigaFlops = (flopsPerMMul * 1.0e-9f) /
	(msecPerKernelExecution / 1000.0f);

	printf("GFlops: %lf\n", gigaFlops);
	printf("TPKernel: %lf\n", msecPerKernelExecution);
	printf("Size: %d\n", N);

	matrixMultCPU(a,b,d);

	hipMemcpy(c, dev_c, size, hipMemcpyDeviceToHost);
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);

// comprobando
	for (int y = 0; y < N; y++) {
		for (int x = 0; x < N; x++) {
			if (c[y][x] != d[y][x]){
				printf("ERROR en %d %d, %d != %d\n", y,x,c[y][x], d[y][x]);
				return 1;
			}
		}
	}
	printf("SUCCESS\n");
	return 0;
}