#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>
void add(int n, float *x, float *y) {
    
    for (int i =0; i < n; i++ ){
        y[i]=x[i]+y[i];
    }
}
Writing suma0.cu
int main(void) {
    
    int N = 1 <<20;  // N = 2^20 = 1024*1024= 1.048.576
    float *x = new float[N];
    float *y = new float[N]; 
    
    for (int i =0; i < N; i++ ){
        x[i]= 1.0f;
        y[i]= 2.0f;
    }
    add(N, x, y);
   float maxError = 0.0f;
   int contError = 0;
   
   for (int i=0; i <N; i++){
       maxError=fmax(maxError,fabs(y[i]-3.0f));
       if (y[i] != 3.0) contError++; 
   }
   std::cout << "suma de " << N << " Elementos" << std::endl;
   std::cout << "Número de Errores: " <<contError << std::endl;
   std::cout << "Max error: " <<maxError << std::endl;
   
   delete [] x;
   delete [] y;
   return 0;
}
